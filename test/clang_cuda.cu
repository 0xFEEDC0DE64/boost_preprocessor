# /* **************************************************************************
#  *                                                                          *
#  *     (C) Copyright Edward Diener 2019.
#  *     Distributed under the Boost Software License, Version 1.0. (See
#  *     accompanying file LICENSE_1_0.txt or copy at
#  *     http://www.boost.org/LICENSE_1_0.txt)
#  *                                                                          *
#  ************************************************************************** */
#
# /* See http://www.boost.org for most recent version. */
#
# include <boost/preprocessor/cat.hpp>
#
# define BEGIN typedef int BOOST_PP_CAT(test_, __LINE__)[((
# define END )==1) ? 1 : -1];

#if defined(__clang__) && defined(__HIPCC__) && defined(__CUDA__)

BEGIN BOOST_PP_VARIADICS == 1 END

#else

BEGIN 0 == 1 END

#endif

int main(void) {
    return 0;
}
